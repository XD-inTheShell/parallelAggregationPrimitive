#include "hip/hip_runtime.h"
// Hash function adapts and modifies based on 
// https://github.com/nosferalatu/SimpleGPUHashTable/tree/master
// which is lock free insert, but do not ensure the modification of
// value thread safe.

#include "basichash.cuh"
#include "../common.h"
#include <stdlib.h>
#include <stdio.h>

// The hashmaps in this implementation assumes that hash key can 
// not be erased.
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity-1);
}
// Local Hashtable ------------------------------
__device__ void localhashUpdate(KeyValue* privateHashtable,
                                 uint32_t key, uint32_t value){
    uint32_t slot = hash(key);
    while (true)
    {
        // Insert myself
        if (privateHashtable[slot].key==kEmpty){
            privateHashtable[slot].key = key;
            privateHashtable[slot].value = value;
            return;
        } 
        // Insertion failed, check if this is my slot 
        else if(privateHashtable[slot].key==key){
            privateHashtable[slot].value += value;
            return;
        }   
        slot = (slot + 1) & (kHashTableCapacity-1);
    }
}
__global__ void localhashAggregate(KeyValue* globalHashtable,
                            Key * device_keys, Value * device_values,
                            long unsigned int cap, long unsigned int base, 
                            unsigned int step, unsigned int const launch_thread){

    KeyValue privateHashtable[KEYSIZE];
    for(int i=0; i<KEYSIZE; i++){
        privateHashtable[i].key = kEmpty;
        // privateHashtable[i].value = vEmpty;
    }

    long unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    for(unsigned int i=index; i<index+step*launch_thread; i+=launch_thread){
        if((i+base) < cap){
            Key key     = device_keys[i];
            Value value = device_values[i];
            localhashUpdate(privateHashtable, key, value);
        }
    }

    // Write to global hash
    for(int i=0; i<KEYSIZE; i++){
        Key key = privateHashtable[i].key;
        if(key!=kEmpty){
            Value value = privateHashtable[i].value;
            hashtable_update(globalHashtable, key, value);
        }
    }
}
// Global Hashtable -----------------------------
__device__ void atomicAddValue(KeyValue* hashtable, uint32_t slot, Value value){
    // hashtable[slot].value = 1111;
    Value prevv = hashtable[slot].value;
    Value writev = prevv + value;
    while(atomicCAS(&hashtable[slot].value, prevv, writev)!=prevv){
        prevv = hashtable[slot].value;
        writev = prevv + value;
    }
    return;
}

// todo: handle VALUE = EMPTYVALUESENTINEL
__device__  __inline__ void hashtable_update(KeyValue* hashtable, Key key, Value value)
{
    uint32_t slot = hash(key);

    while (true)
    {
        Key prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
        if (prev == kEmpty)
        {
            Value prevv = atomicCAS(&hashtable[slot].value, vEmpty, value);
            // No thread gets before me
            if(prevv == vEmpty){
                return;
            }
            // Some thread with the same key gets before me and wrote its value
            // I need to add my value to its value
            atomicAddValue(hashtable, slot, value);
            // Function only returns if it succesfully added my value, safe to return.
            return; 
        } else if(prev == key) {
            // Some other thread with the same key inserted,
            // since we share the same key, I need to atomically add mine.
            atomicAddValue(hashtable, slot, value);
            return;
        }

        slot = (slot + 1) & (kHashTableCapacity-1);
    }
    
}

__global__ void hashtable_empty(KeyValue* hashtable){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < kHashTableCapacity){
        hashtable[index].key    = kEmpty;
        hashtable[index].value  = vEmpty;
    }

    return;
}

void print_hashtable(KeyValue* device_hashtable, KeyValue* host_hashtable){
    hipMemcpy(host_hashtable, device_hashtable, sizeof(KeyValue) * kHashTableCapacity,
               hipMemcpyDeviceToHost);
    for(int i=0; i<kHashTableCapacity; i++){
        printf("entry %d: \tkey=%x, value=%x\n", i, host_hashtable[i].key, host_hashtable[i].value);
    }
}

void export_hashtable(KeyValue* device_hashtable, KeyValue* host_hashtable, std::unordered_map<Key, Value> &umap){
    hipMemcpy(host_hashtable, device_hashtable, sizeof(KeyValue) * kHashTableCapacity,
               hipMemcpyDeviceToHost);
    for(int i=0; i<kHashTableCapacity; i++){
        if(host_hashtable[i].key!=kEmpty){
            // printf("entry %d: \tkey=%u, value=%u\n", i, host_hashtable[i].key, host_hashtable[i].value);
            umap[host_hashtable[i].key]=host_hashtable[i].value;
        }
        
    }
}

// Host create hashtable
KeyValue* create_hashtable() 
{
    // Allocate memory
    KeyValue* hashtable;
    hipMalloc(&hashtable, sizeof(KeyValue) * kHashTableCapacity);

    // Initialize hash table to empty
    // Since we have a specific pattern we want to set, use a kernel to set it.
    const int threadsPerBlock = 512;
    const int blocks = (kHashTableCapacity + threadsPerBlock - 1) / threadsPerBlock;
    hashtable_empty<<<blocks, threadsPerBlock>>>(hashtable);
    // static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    // hipMemset(hashtable, 0xff, sizeof(KeyValue) * kHashTableCapacity);

    return hashtable;
}

__global__ void simplehashAggregateKernel(KeyValue* hashtable, 
                            Key * device_keys, Value * device_values,
                            long unsigned int cap, long unsigned int base, 
                            unsigned int step, unsigned int const launch_thread){
    long unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    long unsigned int start = index * launch_thread;
    long unsigned int offset = index + base;
    // Key key     = device_keys[index];
    //         Value value = device_values[index];
    //         hashtable_update(hashtable, key, index);
    for(unsigned int i=index; i<index+step*launch_thread; i+=launch_thread){
        if((i+base) < cap){
            Key key     = device_keys[i];
            Value value = device_values[i];
            hashtable_update(hashtable, key, value);
        }
    }
    

}
// __device__ bool void hashtable_lookup(KeyValue* hashtable, KeyValue* kvs, unsigned int numkvs)
// {
//     unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (threadid < numkvs)
//     {
//         uint32_t key = kvs[threadid].key;
//         uint32_t slot = hash(key);

//         while (true)
//         {
//             if (hashtable[slot].key == key)
//             {
//                 kvs[threadid].value = hashtable[slot].value;
//                 return;
//             }
//             if (hashtable[slot].key == kEmpty)
//             {
//                 kvs[threadid].value = kEmpty;
//                 return;
//             }
//             slot = (slot + 1) & (kHashTableCapacity - 1);
//         }
//     }
// }